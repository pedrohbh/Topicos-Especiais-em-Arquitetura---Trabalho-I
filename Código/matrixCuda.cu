#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <time.h>

int qtdite = 1;

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>


typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
} Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// tamanho do bloco de thread
#define BLOCK_SIZE 32

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}



// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;

    d_A.width  = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);

     hipError_t error;
    error = hipMalloc(&d_A.elements, size);

    //testa se a matriz conseguiu ser alocada na memoria
    if (error != hipSuccess)
    {
        printf("falha ao alocar a memoria da matriz. (code %d), line(%d)\n",  error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("falha ao copia a matriz para a GPU  (code %d), line(%d)\n", error,  __LINE__);
        exit(EXIT_FAILURE);
    }

    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);

    error = hipMalloc(&d_B.elements, size);
    if (error != hipSuccess)
    {
        printf("erro ao locar a memoria na GPU  (code %d), line(%d)\n", error,  __LINE__);
        exit(EXIT_FAILURE);
    }

    error= hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("falha ao alocar a memoria da matriz  (code %d), line(%d)\n", error,  __LINE__);
        exit(EXIT_FAILURE);
    }

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    // O tipo dim3 'e usado pelo Cuda para definir a estrutura das threads e dos blocos

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE,1);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

    /// declara as avriaveis para medir o tempo em CUDA
    hipEvent_t start,stop;

    //declara variaveis do tempo em C
//    clock_t tinicial,tfinal;

    error = hipEventCreate(&start);
    error = hipEventCreate(&stop);
    if (error != hipSuccess)
    {
        printf("Erro ao startar o evento\n");
        exit(EXIT_FAILURE);
    }

    //inicia
    error = hipEventRecord(start, NULL);

//    tinicial = clock();
	#pragma omp parallel for private( t ) shared( d_A, d_B, d_C, qtdite, dimGrid, dimBlock )
    for (int t=0; t<qtdite; ++t)
    {
      // printf("executando a iteracao %d\n",t);
       MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    }


    // Wait for the stop event to complete

    error = hipEventRecord(stop, NULL);

    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        printf("erro ao startar o evento\n");
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);
//    tfinal = clock();
//    double tempoemC = (double)((tfinal - tinicial) / CLOCKS_PER_SEC);
//    double tempoemC = (double)((tfinal - tinicial) );

    error = hipEventDestroy(start);
    error = hipEventDestroy(stop);


// Compute and print the performance
    double  msectotal        = msecTotal;
    double msecPerMatrixMul  = msecTotal /qtdite;
    double flopsPerMatrixMul = 2.0 * (double)A.height * (double)A.width * (double)B.height;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("\e[H\e[2J");
    printf("                        ************ Resultados *****************\n\n");
    printf("Tamanho da matriz utilizada.............= (%i) x (%i) \n",d_A.width,d_A.width);
    printf("Tamanho do bloco theads e ladrilho......= (32 x 32)\n");
    printf("Numero de iteracoes executadas..........= %i \n\n",qtdite);
    printf("Performance ............................= %.2f GFlop/s \n",gigaFlops);
    printf("Tempo Total (Pelo hipEventElapsedTime) = %.4f segundos \n\n\n",msectotal/1000);



    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results

//#pragma unroll 2

    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) 
      {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // para esperar todas as threads terminarem o processamento
        __syncthreads();

        // Multiply Asub and Bsub together

//#pragma unroll 
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

/**
 * Program main
 */

int main(int argc, char **argv)
{
   Matrix A, B, C;
 
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        // usa sempre matrizes quadraticas do mesmo tamanho
        printf("Use   -pla= \n");
        printf("      -tam= tamanho da matriz\n");
        printf("      -qtd= qtd de interacao\n");
        exit(EXIT_SUCCESS);
    }

    // define os valores default
    int tammatriz = 512; //tamanho padrao

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("\n\nGPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // tamanho matrizes
    if (checkCmdLineFlag(argc, (const char **)argv, "tam"))
    {
      tammatriz = getCmdLineArgumentInt(argc, (const char **)argv, "tam");
    }

    
    if (checkCmdLineFlag(argc, (const char **)argv, "qtd"))
    {
      qtdite = getCmdLineArgumentInt(argc, (const char **)argv, "qtd");
    }


   A.height = tammatriz;
   A.width  = tammatriz;
   A.elements = (float*)malloc(A.width * A.height * sizeof(float));

    //testa se a matriz conseguiu ser alocada na memoria do host
//    if (A = NULL)
//    {
//        fprintf(stderr, "Failed to allocate host matrix C!\n");
//        exit(EXIT_FAILURE);
//    }

   B.height = tammatriz;
   B.width = tammatriz;
   B.elements = (float*)malloc(B.width * B.height * sizeof(float));
   C.height = A.height;
   C.width = B.width;
   C.elements = (float*)malloc(C.width * C.height * sizeof(float));


//   preenche a matriz
   for(int i = 0; i < A.height; i++)
   {
     for(int j = 0; j < A.width; j++)
        A.elements[i*A.width + j] = -0.01f;
   }

   for(int i = 0; i < B.height; i++)
   {   for(int j = 0; j < B.width; j++)
        B.elements[i*B.width + j] = 0.02f;
   }


   MatMul(A, B, C);


//   for(int i = 0; i <  A.height; i++)
//   {
//    for(int j = 0; j <  A.width; j++)
//      printf("%f ", A.elements[i*A.width + j]);
//      printf("\n");
//   }

//   printf("\n");
//   for(int i = 0; i <  B.height; i++)
//   {
//     for(int j = 0; j < B.width; j++)
//      printf("%f ", B.elements[i*B.width + j]);
//      printf("\n");
//    }
//   printf("\n");

//   for(int i = 0; i <  C.height; i++)
//   {
//    for(int j = 0; j <  C.width; j++)
//     printf("%f ", C.elements[i*C.width + j]);
//      printf("\n");
//    }
//     printf("\n");


}
